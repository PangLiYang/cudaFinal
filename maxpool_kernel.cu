
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <chrono>
#include <algorithm>
#include <cmath>

#define timestamp(__var__) auto __var__ = std::chrono::system_clock::now();
inline double getDuration(std::chrono::time_point<std::chrono::system_clock> a,
                          std::chrono::time_point<std::chrono::system_clock> b)
{
    return std::chrono::duration<double>(b - a).count();
}

using namespace std;

const int THREADS_PER_BLOCK = 32;
//const int N = 232960 >> 8 << 8;
//const int N = 716847 >> 8 << 8;
const int N = 2449029 >> 8 << 8;
//const int N = 4096;

const int dim_in = 256, dim_out = 64;

__global__ void maxpool(float *, float *, unsigned int *);

int main() {

    cout<<"N = "<< N << ", dim_in = " << dim_in << ", dim_out = " << dim_out << ", preparing data..." << endl;

    float *data, *value;
    unsigned int *indices;

    hipMallocManaged(&data,  N * dim_in  * sizeof(float));
    hipMallocManaged(&value, N * dim_out * sizeof(float));
    hipMallocManaged(&indices, N * dim_out * sizeof(unsigned int));

    default_random_engine engine;
    engine.seed(123);

    uniform_real_distribution<float> rd(0, 1);

    generate(data, data + N * dim_in, [&](){ return rd(engine); });

    unsigned int shared_mem_size = THREADS_PER_BLOCK * dim_in * sizeof(float);

    cout<<"Config GridDim = "<< N / THREADS_PER_BLOCK << ", BlockDim = " << THREADS_PER_BLOCK << ", shared_mem_size = " << shared_mem_size << endl;

    dim3 grid(N / THREADS_PER_BLOCK, 1, 1);
    dim3 block(THREADS_PER_BLOCK, 1, 1);

    int times = 100;
    for (int i = 0; i < times; i++) {
        maxpool <<< grid, block, shared_mem_size >>> (data, value, indices);
    }

    hipDeviceSynchronize();
    double measured_time = 0;

    for (int i = 0; i < times; i++) {
        timestamp(t0);
        maxpool <<< grid, block, shared_mem_size >>> (data, value, indices);
        hipDeviceSynchronize();
        timestamp(t1);
        measured_time += getDuration(t0, t1);
    }

    cout << "max-pooling time = " << measured_time / times * 1000 << " ms" <<endl;

    hipDeviceSynchronize();

    for (int i = 0; i < 64; i += 1) {
        cout << "value[" << i << "] = " << value[i] << endl;
    }

    for (int i = 0; i < 64; i += 1) {
        cout << "indices[" << i << "] = " << indices[i] << endl;
    }

    hipFree(data);
    hipFree(value);
    hipFree(indices);

    return 0;
}

__global__ void maxpool(float *data, float *value, unsigned int *indices) {

    extern __shared__ float buffer[];

    const int sqrt_dim_in = 16;
    const int thread_offset = threadIdx.x * dim_in;
    const int block_offset = blockIdx.x * THREADS_PER_BLOCK * dim_in;

#pragma unroll
    for (unsigned int i = 0; i < dim_in; i += 1) {
        buffer[thread_offset + i] = data[block_offset + thread_offset + i];
    }

    //__syncwarp();

    float v;
    int pos;
    int offset = 0;

#pragma unroll
    for (int xx = 0; xx < sqrt_dim_in; xx += 2) {

        for (int yy = 0; yy < sqrt_dim_in; yy += 2) {

            pos = xx * sqrt_dim_in + yy;
            v = buffer[thread_offset + pos];

            if (buffer[thread_offset + xx * sqrt_dim_in + yy + 1] > v) {
                pos = xx * sqrt_dim_in + yy + 1;
                v = buffer[thread_offset + pos];
            }

            if (buffer[thread_offset + (xx + 1) * sqrt_dim_in + yy] > v) {
                pos = (xx + 1) * sqrt_dim_in + yy;
                v = buffer[thread_offset + pos];
            }

            if (buffer[thread_offset + (xx + 1) * sqrt_dim_in + yy + 1] > v) {
                pos = (xx + 1) * sqrt_dim_in + yy + 1;
                v = buffer[thread_offset + pos];
            }

            value[blockIdx.x * THREADS_PER_BLOCK * dim_out + threadIdx.x * dim_out + offset] = v;
            indices[blockIdx.x * THREADS_PER_BLOCK * dim_out + threadIdx.x * dim_out + offset] = pos;

            offset += 1;

        }
    }
}